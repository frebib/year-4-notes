/*
 * Name:    Joe Groocock
 * ID:      1467414
 *
 * Goals:
 *   - Block scan       ✓
 *   - Full scan        ✓
 *   - BCAO             ✓
 *
 * Performance:
 *   - Block w/o BCAO   0.61907 ms
 *   - Block w/  BCAO   0.44096 ms
 *   - Full  w/o BCAO   1.04813 ms
 *   - Full  w/  BCAO   0.85840 ms
 *
 * Hardware:
 *   - CPU:     i5 4690K @ 4.0Ghz
 *   - GPU:     GTX 1070 @ 1.8Ghz
 *
 * Optimisations:
 *   - Using the constant BLOCK_SIZE macro instead of a variable read from the
 *     user or calculated allows the compiler to inline many computations at
 *     compile-time and unroll loops through constant propogation. This gives a
 *     ~30% performance improvement at runtime.
 *   - Using the smallest possible BLOCK_SIZE with a level-3 scan in most cases
 *     is marginally faster than a fixed value of 1024 (/whatever the maximum is
 *     for the hardware) so is preferred.
 *   - Often running multiple scans back-to-back or running the process multiple
 *     times reduces the runtime slightly. I'm putting this down to GPU cache
 *     utilisation, most likely.
 */

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <fcntl.h>
#include <time.h>
#include <sys/param.h>

#include <hip/hip_runtime.h>

///////////////////
// CONFIGURATION //
///////////////////

/* Define BCAO to enable 'Bank Conflict Avoidance Optimization' */
#define BCAO 1
/* Define MULTILEVEL to enable L2/L3 scans and blockadd */
#define MULTILEVEL 1
#define GPU_SINGLETHREAD 1
#define BLOCK_SIZE 256

//////////////////
//////////////////

//#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) \
    (((n) >> LOG_NUM_BANKS) + ((n) >> (2 * LOG_NUM_BANKS)))


/*
 * A helper macro to simplify handling cuda error checking
 */
#define cudaErr(e, msg) { \
    hipError_t err = e; \
    if (err != hipSuccess) { \
        printf("Error: %s: %s <%s:%d@%s>\n", msg, hipGetErrorString(err), \
                __FILE__, __LINE__, __func__); \
        exit(EXIT_FAILURE); \
    } \
}

#define printArr(pre, arr, len, _ofs, _count) { \
    uint32_t ofs = MAX(0, _ofs); \
    uint32_t count = MIN(_count, len - ofs); \
    printf("%s%s", pre, (ofs == 0 ? "[" : "[.. ")); \
    for(uint acc = ofs; acc < ofs + count; acc++) \
        printf("%2d, ", arr[acc]); \
    printf("%2d%s\n", arr[ofs + count], (ofs + count >= len ? "]" : " ..]")); \
}

#define timespec_diff(a, b, out) { \
    if (((b)->tv_nsec - (a)->tv_nsec) < 0) { \
        (out)->tv_sec = (b)->tv_sec - (a)->tv_sec - 1; \
        (out)->tv_nsec = (b)->tv_nsec - (a)->tv_nsec + 1000000000; \
    } else { \
        (out)->tv_sec = (b)->tv_sec - (a)->tv_sec; \
        (out)->tv_nsec = (b)->tv_nsec - (a)->tv_nsec; \
    } \
}

#define scancmp(msg, ref, out, len, ms, extra) { \
    long idx = arrcmp((ref), (out), sizeof(int), (len)); \
    if (idx > 0) { \
        printf("    - [!] " msg " differs at index %zu (%.5f ms)\n", idx - 1, (ms)); \
        extra \
        /* Print partial array section for debugging */ \
        printArr("      - ref: ", ref, len, idx - 4, 12); \
        printArr("        gpu: ", out, len, idx - 4, 12); \
    } else { \
        printf("    - [✓] " msg " success (%.5f ms)\n", (ms)); \
    } \
}

/**
 * Compares two arrays for equality
 * @param arr array to compare
 * @param brr array to compare
 * @param elem_sz size of each array element (in bytes)
 * @param len length of the smallest array, in elements (not bytes)
 * @return 0 when arr and brr are identical otherwise nth element where they
 *         first differ (index + 1)
 */
uint32_t arrcmp(void *arr, void *brr, uint32_t elem_sz, uint32_t len) {
    for (uint32_t i = 0; i < len; i++) {
        uint32_t idx = i * elem_sz;
        if (memcmp((uint8_t *) arr + idx, (uint8_t *) brr + idx, elem_sz) != 0)
            return i + 1;
    }
    return 0;
}


// tmp[] is local to the block, not global to the array
extern __shared__ int tmp[];
__global__ void prescan(int *out, int *in, int *last, uint32_t count) {

    uint32_t thid = threadIdx.x;
    uint32_t thid2 = thid << 1;
    uint32_t offset = 1;

    // Offset of block within (full) array
    uint32_t blockOfs = blockIdx.x * BLOCK_SIZE;

#ifdef BCAO
    uint32_t ai = thid;
    uint32_t bi = thid + (BLOCK_SIZE >> 1);
    uint32_t bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    uint32_t bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    tmp[ai + bankOffsetA] = in[ai + blockOfs];
    tmp[bi + bankOffsetB] = in[bi + blockOfs];
#else
    tmp[thid2]     = in[blockOfs + thid2];
    tmp[thid2 + 1] = in[blockOfs + thid2 + 1];
#endif

    // build sum in place up the tree
    for (uint32_t d = BLOCK_SIZE >> 1; d > 0; d >>= 1) {
        __syncthreads();

        // Ensure we stay within the bounds of the data size
        if (thid < d) {
            uint32_t aii = offset * (thid2 + 1) - 1;
            uint32_t bii = offset * (thid2 + 2) - 1;
#ifdef BCAO
            aii += CONFLICT_FREE_OFFSET(aii);
            bii += CONFLICT_FREE_OFFSET(bii);
#endif
            tmp[bii] += tmp[aii];
        }

        offset <<= 1;
    }

    if (thid == 0) {
#ifdef BCAO
        uint32_t lastIdx = BLOCK_SIZE - 1 + CONFLICT_FREE_OFFSET(BLOCK_SIZE - 1);
#else
        uint32_t lastIdx = BLOCK_SIZE - 1;
#endif

        if (last != NULL) {
            // save the last element before clearing it
            last[blockIdx.x] = tmp[lastIdx];
        }
        // clear the last element
        tmp[lastIdx] = 0;
    }

    // traverse down tree & build scan
    for (uint32_t d = 1; d < BLOCK_SIZE; d <<= 1) {
        offset >>= 1;
        __syncthreads();

        if (thid < d) {
            uint32_t aii = offset * (thid2 + 1) - 1;
            uint32_t bii = offset * (thid2 + 2) - 1;
#ifdef BCAO
            aii += CONFLICT_FREE_OFFSET(aii);
            bii += CONFLICT_FREE_OFFSET(bii);
#endif

            int t = tmp[aii];
            tmp[aii] = tmp[bii];
            tmp[bii] += t;
        }
    }
    __syncthreads();

    // write results to device memory
#ifdef BCAO
    out[ai + blockOfs] = tmp[ai + bankOffsetA];
    out[bi + blockOfs] = tmp[bi + bankOffsetB];
#else
    out[blockOfs + thid2]     = tmp[thid2];
    out[blockOfs + thid2 + 1] = tmp[thid2 + 1];
#endif
}

__global__ void blockadd(int *arr, uint32_t len, int *sums) {
    uint32_t dstIdx = threadIdx.x + blockDim.x * blockIdx.x;
    if (dstIdx < len) {
        arr[dstIdx] += sums[blockIdx.x];
    }
}

void blockscan_gpu(int *arr, uint32_t len, int *out, float *ms) {

    // Length in bytes
    uint32_t bytLen = len * sizeof(int);
    uint32_t shm_sz = (BLOCK_SIZE * 4) * sizeof(int);
    // Number of blocks [ ceil(tc->len / BLOCK_SIZE) ]
    uint32_t grid_sz = 1 + (len - 1) / (BLOCK_SIZE);
    uint32_t l3_grdsz = 1 + (grid_sz - 1) / (BLOCK_SIZE);

    int *gpuIn = NULL,    *gpuOut = NULL,
        *gpuSums = NULL,  *gpuIncs = NULL,
        *gpuSums2 = NULL, *gpuIncs2 = NULL;

    cudaErr(hipMalloc(&gpuIn, bytLen), "hipMalloc");
    cudaErr(hipMalloc(&gpuOut, bytLen), "hipMalloc");
    cudaErr(hipMalloc(&gpuSums, grid_sz * sizeof(int)), "hipMalloc");
#ifdef MULTILEVEL
    cudaErr(hipMalloc(&gpuIncs, grid_sz * sizeof(int)), "hipMalloc");
    cudaErr(hipMalloc(&gpuSums2, l3_grdsz * sizeof(int)), "hipMalloc");
    cudaErr(hipMalloc(&gpuIncs2, l3_grdsz * sizeof(int)), "hipMalloc");
#endif

    // Create Device timer event objects
    hipEvent_t start, stop;
    cudaErr(hipEventCreate(&start), "hipEventCreate");
    cudaErr(hipEventCreate(&stop), "hipEventCreate");

    // Copy host array to GPU memory
    cudaErr(hipMemcpy(gpuIn, arr, bytLen, hipMemcpyHostToDevice), "hipMemcpy");

    /*
     * Start the timer!
     */
    cudaErr(hipEventRecord(start, 0), "hipEventRecord");

    // <<< blocks-per-grid, threads-per-block, shared-mem(bytes) >>>
    prescan<<<grid_sz, BLOCK_SIZE / 2, shm_sz>>>(gpuOut, gpuIn, gpuSums, len);
    cudaErr(hipGetLastError(), "prescan<<<>>>");

#ifdef MULTILEVEL
    // Check if we need a 2rd scan level
    if (grid_sz > 1) {
        // Layer 2 scan
        prescan<<<l3_grdsz, BLOCK_SIZE / 2, shm_sz>>>(gpuIncs, gpuSums, gpuSums2, grid_sz);
        cudaErr(hipGetLastError(), "prescan<<<>>>");

        // Check if we need a 3rd scan level
        if (l3_grdsz > 1) {
            prescan<<<l3_grdsz, BLOCK_SIZE / 2, shm_sz>>>(gpuIncs2, gpuSums2, NULL, l3_grdsz);
            cudaErr(hipGetLastError(), "prescan<<<>>>");

            blockadd<<<l3_grdsz, BLOCK_SIZE>>>(gpuIncs, grid_sz, gpuIncs2);
            cudaErr(hipGetLastError(), "blockadd<<<>>>");
        }
    }

    blockadd<<<grid_sz, BLOCK_SIZE>>>(gpuOut, len, gpuIncs);
    cudaErr(hipGetLastError(), "blockadd<<<>>>");
#endif

    /*
     * Stop the timer!
     */
    cudaErr(hipEventRecord(stop, 0), "hipEventRecord");
    cudaErr(hipEventSynchronize(stop), "hipEventSynchronize");
    cudaErr(hipDeviceSynchronize(), "hipDeviceSynchronize");
    cudaErr(hipEventElapsedTime(ms, start, stop), "hipEventElapsedTime");

    cudaErr(hipMemcpy(out, gpuOut, bytLen, hipMemcpyDeviceToHost), "hipMemcpy");

    // Deallocate memory
    cudaErr(hipFree(gpuIn), "hipFree");
    cudaErr(hipFree(gpuOut), "hipFree");
    cudaErr(hipFree(gpuSums), "hipFree");
    cudaErr(hipFree(gpuIncs), "hipFree");
    cudaErr(hipFree(gpuSums2), "hipFree");
    cudaErr(hipFree(gpuIncs2), "hipFree");
    cudaErr(hipEventDestroy(start), "hipEventDestroy");
    cudaErr(hipEventDestroy(stop), "hipEventDestroy");
    cudaErr(hipDeviceReset(), "Failed to reset the device");
}

/**
 * Performs a sequential blockscan on the CPU, in a single thread
 */
__host__ void blockscan_cpu(const int *in, int *out, uint32_t count, float *ms) {

    struct timespec start, end, diff;
    clock_gettime(CLOCK_MONOTONIC, &start);

    // Perform sequential scan
    out[0] = 0;
    for (uint64_t i = 1; i < count; i++)
        out[i] = in[i - 1] + out[i - 1];

    clock_gettime(CLOCK_MONOTONIC, &end);
    timespec_diff(&start, &end, &diff);
    *ms = (float) (diff.tv_sec * 1000.0 + diff.tv_nsec / 1000000.0);

}

/**
 * Performs a sequential blockscan on the GPU, in a single thread/block
 * Should be invoked as blockscan_single_gpu<<<1, 1>>>(..)
 */
__global__ void _blockscan_single_gpu(const int *in, int *out, uint32_t count) {
    out[0] = 0;
    for (uint64_t i = 1; i < count; i++)
        out[i] = in[i - 1] + out[i - 1];
}
__host__ void blockscan_single_gpu(const int *in, int *out, uint32_t count,
                                     float *ms) {

    uint32_t bytLen = sizeof(int) * count;

    hipEvent_t gpuStart, gpuStop;
    cudaErr(hipEventCreate(&gpuStart), "hipEventCreate");
    cudaErr(hipEventCreate(&gpuStop), "hipEventCreate");

    // Memory management
    int *gpuIn = NULL, *gpuOut = NULL;
    cudaErr(hipMalloc(&gpuIn, bytLen), "hipMalloc");
    cudaErr(hipMalloc(&gpuOut, bytLen), "hipMalloc");
    cudaErr(hipMemcpy(gpuIn, in, bytLen, hipMemcpyHostToDevice),
            "hipMemcpy");
    cudaErr(hipEventRecord(gpuStart, 0), "hipEventRecord");
    // Timer start

    _blockscan_single_gpu<<<1, 1>>>(gpuIn, gpuOut, count);

    // Timer end
    cudaErr(hipEventRecord(gpuStop, 0), "hipEventRecord");
    cudaErr(hipEventSynchronize(gpuStop), "hipEventSynchronize");
    cudaErr(hipDeviceSynchronize(), "hipDeviceSynchronize");
    cudaErr(hipEventElapsedTime(ms, gpuStart, gpuStop),
            "hipEventElapsedTime");

    cudaErr(hipMemcpy(out, gpuOut, bytLen, hipMemcpyDeviceToHost), "hipMemcpy");

    cudaErr(hipFree(gpuIn), "hipFree");
    cudaErr(hipFree(gpuOut), "hipFree");
    cudaErr(hipDeviceReset(), "Failed to reset the device");
}


/**
 * Host main routine
 */
int main(void) {

    /*
     * Define some test cases
     */
    int devurandom = open("/dev/urandom", O_RDONLY);
    if (devurandom < 0) perror("open");

#define DEFINE_TEST(rnd, name, size) \
    uint32_t name##_sz = (size); \
    int *(name) = (int *) malloc((size) * sizeof(int)); \
    read(rnd, (name), (size) * sizeof(int)); \

    DEFINE_TEST(devurandom, test1, BLOCK_SIZE);
    DEFINE_TEST(devurandom, test2, BLOCK_SIZE << 4);
    DEFINE_TEST(devurandom, test3, 10000000);

    // Fight me @domwillia.ms
    close(devurandom);

    struct test_case {
        int *arr;
        uint32_t len;
    } tests[] = {
        { test1, test1_sz },
        { test2, test2_sz },
        { test3, test3_sz },
    };

    /*
     * Run each test case and print the results
     */
    uint32_t count = sizeof(tests) / sizeof(struct test_case);
    printf("[~] Block size is %u\n", BLOCK_SIZE);
    printf("[#] %u tests to run\n", count);

    for (uint i = 0; i < count; i++) {
        // Compute the simple sequential scan on the CPU
        struct test_case *tc = &tests[i];

        uint32_t bytLen = sizeof(int) * tc->len;
        int *ref = (int *) malloc(bytLen);
        int *out = (int *) malloc(bytLen);

        // Print input test case
        printf("  [*] Testing case #%d, %u elements\n", i + 1, tc->len);

        float refMs = -1, gpuMs = -1;
        /*
         * Compute CPU scan
         */
        blockscan_cpu(tc->arr, ref, tc->len, &refMs);
        printf("    - [✓] CPU reference scan (%.5f ms)\n", refMs);

#ifdef GPU_SINGLETHREAD
        /*
         * Compute GPU single-threaded scan
         */
        blockscan_single_gpu(tc->arr, out, tc->len, &gpuMs);
        scancmp("GPU sequential scan", ref, out, tc->len, gpuMs, {});

#endif
        /*
         * Compute GPU scan
         */
        blockscan_gpu(tc->arr, tc->len, out, &gpuMs);
        scancmp("GPU parallel scan", ref, out, tc->len, gpuMs, {
                uint32_t shm_sz = (BLOCK_SIZE * 4) * sizeof(int);
                uint32_t grid_sz = 1 + (tc->len - 1) / (BLOCK_SIZE);
                printf("      - %u blocks, %u threads, %u tmp bytes\n",
                       grid_sz, BLOCK_SIZE, shm_sz);
        });

        printf("    - Speedup: CPU -> GPU = %.2fx\n", refMs / gpuMs);
        printf("\n");
    }

    // Deallocate dynamically allocated arrays
    for (uint i = 0; i < count; i++)
        free(tests[i].arr);
}
